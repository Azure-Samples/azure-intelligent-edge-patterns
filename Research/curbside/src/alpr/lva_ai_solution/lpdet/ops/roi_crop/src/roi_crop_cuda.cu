#include "hip/hip_runtime.h"
#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <vector>

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600

#else
static __inline__ __device__ double atomicAdd(double *address, double val) {
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;
  if (val==0.0)
    return __longlong_as_double(old);
  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val +__longlong_as_double(assumed)));
  } while (assumed != old);
  return __longlong_as_double(old);
}
#endif

#define CUDA_1D_KERNEL_LOOP(i, n)                            \
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
     i += blockDim.x * gridDim.x)




namespace {

    template <typename scalar_t>
    __global__ void crop_and_resize_forward_kernel(
        const int nthreads,
        const scalar_t* __restrict__ image_ptr,
        const scalar_t* __restrict__ boxes_ptr,
        const int num_boxes,
        const int batch,
        const int image_height,
        const int image_width,
        const int crop_height,
        const int crop_width,
        const int depth,
        const float extrapolation_value,
        scalar_t* crops_ptr
    ) {

        CUDA_1D_KERNEL_LOOP(out_idx, nthreads)
        {
            // NHWC: out_idx = d + depth * (w + crop_width * (h + crop_height * b))
            // NCHW: out_idx = w + crop_width * (h + crop_height * (d + depth * b))
            int idx = out_idx;
            const int x = idx % crop_width;
            idx /= crop_width;
            const int y = idx % crop_height;
            idx /= crop_height;
            const int d = idx % depth;
            const int b = idx / depth;

            const int b_in = static_cast<int>(rintf(boxes_ptr[b * 5 + 0]));
            const scalar_t x1 = boxes_ptr[b * 5 + 1];
            const scalar_t y1 = boxes_ptr[b * 5 + 2];
            const scalar_t x2 = boxes_ptr[b * 5 + 3];
            const scalar_t y2 = boxes_ptr[b * 5 + 4];

            if (b_in < 0 || b_in >= batch)
            {
                continue;
            }

            const scalar_t height_scale =
                (crop_height > 1) ? (y2 - y1) / (crop_height - 1) : 0;
            const scalar_t width_scale =
                (crop_width > 1) ? (x2 - x1) / (crop_width - 1) : 0;

            const scalar_t in_y = (crop_height > 1)
                                    ? y1 + y * height_scale
                                    : 0.5 * (y1 + y2);
            if (in_y < 0 || in_y > image_height - 1)
            {
                crops_ptr[out_idx] = extrapolation_value;
                continue;
            }

            const scalar_t in_x = (crop_width > 1)
                                    ? x1 + x * width_scale
                                    : 0.5 * (x1 + x2);
            if (in_x < 0 || in_x > image_width - 1)
            {
                crops_ptr[out_idx] = extrapolation_value;
                continue;
            }

            const int top_y_index = floorf(in_y);
            const int bottom_y_index = ceilf(in_y);
            const scalar_t y_lerp = in_y - top_y_index;

            const int left_x_index = floorf(in_x);
            const int right_x_index = ceilf(in_x);
            const scalar_t x_lerp = in_x - left_x_index;

            const scalar_t *pimage = image_ptr + (b_in * depth + d) * image_height * image_width;
            const scalar_t top_left = pimage[top_y_index * image_width + left_x_index];
            const scalar_t top_right = pimage[top_y_index * image_width + right_x_index];
            const scalar_t bottom_left = pimage[bottom_y_index * image_width + left_x_index];
            const scalar_t bottom_right = pimage[bottom_y_index * image_width + right_x_index];

            const scalar_t top = top_left + (top_right - top_left) * x_lerp;
            const scalar_t bottom = bottom_left + (bottom_right - bottom_left) * x_lerp;
            crops_ptr[out_idx] = top + (bottom - top) * y_lerp;
        }
    }


    template <typename scalar_t>
    __global__ void crop_and_resize_backward_kernel(
        const int nthreads,
        const scalar_t *grads_ptr,
        const scalar_t *boxes_ptr,
        const int num_boxes,
        const int batch,
        const int image_height,
        const int image_width,
        const int crop_height,
        const int crop_width,
        const int depth,
        scalar_t *grads_image_ptr)
    {
        CUDA_1D_KERNEL_LOOP(out_idx, nthreads)
        {
            // NHWC: out_idx = d + depth * (w + crop_width * (h + crop_height * b))
            // NCHW: out_idx = w + crop_width * (h + crop_height * (d + depth * b))
            int idx = out_idx;
            const int x = idx % crop_width;
            idx /= crop_width;
            const int y = idx % crop_height;
            idx /= crop_height;
            const int d = idx % depth;
            const int b = idx / depth;

            const scalar_t x1 = boxes_ptr[b * 5 + 1];
            const scalar_t y1 = boxes_ptr[b * 5 + 2];
            const scalar_t x2 = boxes_ptr[b * 5 + 3];
            const scalar_t y2 = boxes_ptr[b * 5 + 4];

            const int b_in = static_cast<int>(rintf(boxes_ptr[b * 5]));
            if (b_in < 0 || b_in >= batch)
            {
                continue;
            }

            const scalar_t height_scale =
                (crop_height > 1) ? (y2 - y1) / (crop_height - 1)
                                    : 0;
            const scalar_t width_scale =
                (crop_width > 1) ? (x2 - x1) / (crop_width - 1) : 0;

            const scalar_t in_y = (crop_height > 1)
                                    ? y1 * + y * height_scale
                                    : 0.5 * (y1 + y2);
            if (in_y < 0 || in_y > image_height - 1)
            {
                continue;
            }

            const scalar_t in_x = (crop_width > 1)
                                    ? x1 * (image_width - 1) + x * width_scale
                                    : 0.5 * (x1 + x2) * (image_width - 1);
            if (in_x < 0 || in_x > image_width - 1)
            {
                continue;
            }

            const int top_y_index = floorf(in_y);
            const int bottom_y_index = ceilf(in_y);
            const scalar_t y_lerp = in_y - top_y_index;

            const int left_x_index = floorf(in_x);
            const int right_x_index = ceilf(in_x);
            const scalar_t x_lerp = in_x - left_x_index;

            scalar_t *pimage = grads_image_ptr + (b_in * depth + d) * image_height * image_width;
            const scalar_t dtop = (1 - y_lerp) * grads_ptr[out_idx];
            atomicAdd(
                pimage + top_y_index * image_width + left_x_index,
                (1 - x_lerp) * dtop
            );
            atomicAdd(
                pimage + top_y_index * image_width + right_x_index,
                x_lerp * dtop
            );

            const scalar_t dbottom = y_lerp * grads_ptr[out_idx];
            atomicAdd(
                pimage + bottom_y_index * image_width + left_x_index,
                (1 - x_lerp) * dbottom
            );
            atomicAdd(
                pimage + bottom_y_index * image_width + right_x_index,
                x_lerp * dbottom
            );
        }
    }

}

int crop_and_resize_cuda_forward(
    at::Tensor image,
    at::Tensor rois,
    const float extrapolation_value,
    at::Tensor output)
{

    const int batch = image.size(0);
    const int depth = image.size(1);
    const int image_height = image.size(2);
    const int image_width = image.size(3);
    const int num_boxes = rois.size(0);
    const int crop_height = output.size(2);
    const int crop_width = output.size(3);

    // auto crop = at::zeros(image.type(), {num_boxes, depth, crop_height, crop_width});
    // auto crop = at::zeros({num_boxes, depth, crop_height, crop_width}, image.type());
    const int total_count = num_boxes * crop_height * crop_width * depth;
    const int thread_per_block = 1024;
    const int block_count = (total_count + thread_per_block - 1) / thread_per_block;
    const dim3 blocks(block_count);

    hipError_t err;
    if (total_count > 0){
        AT_DISPATCH_FLOATING_TYPES(image.type(), "crop_and_resize_forward_cuda", ([&] {
            crop_and_resize_forward_kernel<scalar_t><<<blocks, thread_per_block>>>(
                total_count,
                image.data<scalar_t>(),
                rois.data<scalar_t>(),
                num_boxes,
                batch,
                image_height,
                image_width,
                crop_height,
                crop_width,
                depth,
                extrapolation_value,
                output.data<scalar_t>());
        }));

        err = hipGetLastError();

        if (hipSuccess != err)
        {
            fprintf(stderr, "cudaCheckError() failed : %s\n", hipGetErrorString(err));
            exit(-1);
        }
    }

    return 0;
}



int crop_and_resize_cuda_backward(
    at::Tensor grads,
    at::Tensor rois,
    at::Tensor bottom_grads
)
{
    // shape
    const int num_boxes = rois.size(0);
    const int depth = grads.size(1);
    const int crop_height = grads.size(2);
    const int crop_width = grads.size(3);
    const int batch = bottom_grads.size(0);
    const int image_height = bottom_grads.size(2);
    const int image_width = bottom_grads.size(3);
    // n_elements
    //const int image_channel_elements = image_height * image_width;
    //const int image_elements = depth * image_channel_elements;

    //const int channel_elements = crop_height * crop_width;
    //const int crop_elements = depth * channel_elements;

    // init output space
    // auto grads_image = at::zeros(grads.type(), {batch, depth, image_height, image_width});
    // auto grads_image = at::zeros({batch, depth, image_height, image_width}, grads.type());
    const int total_count = num_boxes * crop_height * crop_width * depth;
    const int thread_per_block = 1024;
    const int block_count = (total_count + thread_per_block - 1) / thread_per_block;
    const dim3 blocks(block_count);
    hipError_t err;

    if (total_count > 0)
    {
        AT_DISPATCH_FLOATING_TYPES(grads.type(), "crop_and_resize_backward_cuda", ([&] {
            crop_and_resize_backward_kernel<scalar_t><<<blocks, thread_per_block>>>(
                total_count,
                grads.data<scalar_t>(),
                rois.data<scalar_t>(),
                num_boxes,
                batch,
                image_height,
                image_width,
                crop_height,
                crop_width,
                depth,
                bottom_grads.data<scalar_t>());
            }));

        err = hipGetLastError();
        if (hipSuccess != err)
        {
            fprintf(stderr, "cudaCheckError() failed : %s\n", hipGetErrorString(err));
            exit(-1);
        }
    }

    return 0;

}
